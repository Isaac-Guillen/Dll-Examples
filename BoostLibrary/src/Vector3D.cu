#include "hip/hip_runtime.h"
#include "Vector3D.h"

__host__ __device__ Vector3D::Vector3D(double x, double y, double z)
{
    this->x = x;
    this->y = y;
    this->z = z;
}

__host__ __device__ Vector3D::Vector3D(const Point3D &point)
{
    this->x=point.x;
    this->y=point.y;
    this->z=point.z;
}

__host__ __device__ double Vector3D::Norm()
{
    return sqrt(this->x * this->x + this->y * this->y + this->z * this->z);
}

__host__ void Vector3D::PrintCores()
{
    #pragma omp parallel for
    for(int i = 0; i < 10; i++)
    {
        printf("Hello from thread %d \n", omp_get_thread_num());
    }
}